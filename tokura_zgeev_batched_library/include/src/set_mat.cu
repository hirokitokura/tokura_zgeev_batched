#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>

#include <random>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_complex.h>

#include <tokura_blas.h>


void set_mat(hipDoubleComplex* mat, const int matrix_size, const int mat_num)
{
	int i = 0;
	int j, k;

        std::random_device seed_gen;
        std::mt19937 engine(seed_gen());
	std::uniform_real_distribution<> dist1(-1.0, 1.0);

	for (k = 0; k < mat_num; k++)
	{
		for (i = 0; i < matrix_size; i++)
		{
			for (j = 0; j < matrix_size; j++)
			{
				mat[(j * matrix_size + i) + k * matrix_size * matrix_size].x = dist1(engine);
				mat[(j * matrix_size + i) + k * matrix_size * matrix_size].y = dist1(engine);
			}
		}
	}
	return;
}
